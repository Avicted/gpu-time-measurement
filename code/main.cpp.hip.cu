#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Inspiration and code snippets borrowed from my lecturer Doctor Jan Westerholm at AAU.
// Victor Anderssén 2022 Fall

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <inttypes.h>

// Cpp spec does shenanigans with the usage of <static>, let's be explicit in what
// we mean with 'static'
#define internal static
#define local_persist static
#define global_variable static

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

/* @Note(Victor):
 * All thread/kernel receives these four params as the dim3 type
 *
 * gridDim : gridDim.x, gridDim.y, gridDim.z
 * blockIdx : blockIdx.x, blockIdx.y, blockIdx.z
 * blockDim : blockDim.x, blockDim.y, blockDim.z
 * threadIdx: threadIdx.x, threadIdx.y, threadIdx.z */
__global__ void measure_kernel_memory_transfer_overhead_kernel()
{
}

internal int
get_device()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("   Found %d CUDA devices\n", deviceCount);

    if (deviceCount < 0 || deviceCount > 128)
    {
        return (-1);
    }

    int device;
    for (device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("         Device %s                    = device %d\n", deviceProp.name, device);
        printf("         compute capability           =         %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("         totalGlobalMemory            =        %.2lf GB\n", deviceProp.totalGlobalMem / 1000000000.0);
        printf("         l2CacheSize                  =    %8d B\n", deviceProp.l2CacheSize);
        printf("         regsPerBlock                 =    %8d\n", deviceProp.regsPerBlock);
        printf("         multiProcessorCount          =    %8d\n", deviceProp.multiProcessorCount);
        printf("         maxThreadsPerMultiprocessor  =    %8d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("         sharedMemPerBlock            =    %8d B\n", (int)deviceProp.sharedMemPerBlock);
        printf("         warpSize                     =    %8d\n", deviceProp.warpSize);
        printf("         clockRate                    =    %8.2lf MHz\n", deviceProp.clockRate / 1000.0);
        printf("         maxThreadsPerBlock           =    %8d\n", deviceProp.maxThreadsPerBlock);
        printf("         maxGridSize                  =    %d x %d x %d\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("         maxThreadsDim                =    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    }

    hipSetDevice(0);
    hipGetDevice(&device);

    if (device != 0)
    {
        printf("   Unable to set device 0, using %d instead", device);
    }
    else
    {
        printf("   Using CUDA device %d\n\n", device);
    }

    return (0);
}

int main(int argc, char *argv[])
{
    printf("        Starting the program\n");

    get_device();

    struct timeval st, et;
    struct timezone _tzone;
    const unsigned long N = 1L;
    gettimeofday(&st, &_tzone);

    // Main body
    dim3 threadsInBlock(32, 32);
    dim3 blocksInGrid = dim3(
        ceil((N + threadsInBlock.x - 1) / threadsInBlock.x),
        ceil((N + threadsInBlock.y - 1) / threadsInBlock.y));

    printf("====================================================================\n");
    printf("blocksInGrid:\t{%d, %d, %d} blocks.\nthreadsInBlock:\t%d threads.\n",
           blocksInGrid.x, blocksInGrid.y, blocksInGrid.z, threadsInBlock.x * threadsInBlock.y * threadsInBlock.z);

    const long int number_of_threads = (long int)(threadsInBlock.x * ((long)(threadsInBlock.y)) * threadsInBlock.z * ((blocksInGrid.x * blocksInGrid.y) * blocksInGrid.z));

    printf("number of threads: %ld\n", number_of_threads);

    // Call the GPU kernel(s)
    measure_kernel_memory_transfer_overhead_kernel<<<blocksInGrid, threadsInBlock>>>();

    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
    // Main body end

    gettimeofday(&et, &_tzone);

    int elapsed = ((et.tv_sec - st.tv_sec) * 1000000) + (et.tv_usec - st.tv_usec);
    printf("        The program took %d microseconds\n", elapsed);
    printf("        The program took %d milliseconds\n", elapsed / 1000);
    printf("        The program took %f seconds\n", double((double)elapsed / 1000000.0));
    printf("        To execute the GPU kernel\n");

    return (0);
}
